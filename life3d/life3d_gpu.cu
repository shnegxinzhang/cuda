/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SC24219042
 * 姓名:张胜欣
 * 邮箱:zhang2048525561@163.com
 ------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
using namespace std;

//全局变量，交换指针
char * tmp;
#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
#define EXCHANGE_PTR(x,y) {tmp=x; x=y; y=tmp;}

//using std::cin, std::cout, std::endl;
//using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// 核心计算代码，将世界向前推进T个时刻
void life3d_run_cpu(int N, char *universe, int T)
{
    char *next = (char *)malloc(N * N * N);
    for (int t = 0; t < T; t++)
    {
        // outerloop: iter universe
        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++)
                for (int z = 0; z < N; z++)
                {
                    // inner loop: stencil
                    int alive = 0;
                    for (int dx = -1; dx <= 1; dx++)
                        for (int dy = -1; dy <= 1; dy++)
                            for (int dz = -1; dz <= 1; dz++)
                            {
                                if (dx == 0 && dy == 0 && dz == 0)
                                    continue;
                                int nx = (x + dx + N) % N;
                                int ny = (y + dy + N) % N;
                                int nz = (z + dz + N) % N;
                                alive += AT(nx, ny, nz);
                            }
                    if (AT(x, y, z) && (alive < 5 || alive > 7))
                        next[x * N * N + y * N + z] = 0;
                    else if (!AT(x, y, z) && alive == 6)
                        next[x * N * N + y * N + z] = 1;
                    else
                        next[x * N * N + y * N + z] = AT(x, y, z);
                }
        memcpy(universe, next, N * N * N);
    }
    free(next);
}


// 读取输入文件
// input_file 输入文件的路径
// buffer 存储读取数据的缓冲区
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
// output_file 输出文件的路径
// buffer 存储要写入数据的缓冲区
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
//cpu
    read_file(input_file, universe);
    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run_cpu(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "CPU:" << endl;
    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
