/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SC24219042
 * 姓名:张胜欣
 * 邮箱:zhang2048525561@163.com
 ------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
using namespace std;

//全局变量，交换指针
char * tmp;
#define AT(x, y, z) grid[(x) * N * N + (y) * N + z]
#define EXCHANGE_PTR(x,y) {tmp=x; x=y; y=tmp;}

//using std::cin, std::cout, std::endl;
//using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char *grid)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += grid[i];
    return result;
}

// 打印世界状态
void print_grid(int N, char *grid)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, grid) << endl;
}

// CUDA 核函数，执行 3D 生命游戏的更新规则
__global__ void life3d_kernel(int N, char* grid, char* nextGrid) {
// 计算当前线程在三维网格中的位置
    int x = (blockIdx.x * blockDim.x + threadIdx.x);
    int y = (blockIdx.y * blockDim.y + threadIdx.y);
    int z = (blockIdx.z * blockDim.z + threadIdx.z);
// 若超出网格范围则不处理
    if (x >= N || y >= N || z >= N)
        return;

    int dx,dy,dz;
    int nx,ny,nz,idx;
    // 初始化存储存活邻居的数量alive_nei_num
    int alive_nei_num = 0;
    // 遍历周围 27 个邻居（包括自身，但自身会被跳过）
    for (dx = -1; dx <= 1; dx++) {
        for (dy = -1; dy <= 1; dy++) {
            for (dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0)
                    continue;
                // 考虑边界循环，使用取模操作
                nx = (x + dx + N) % N;
                ny = (y + dy + N) % N;
                nz = (z + dz + N) % N;
                // 统计存活邻居的数量
                alive_nei_num += AT(nx, ny, nz);
            }
        }
    }
    // 计算当前元素在一维数组中的索引
    idx = x * N * N + y * N + z;
    // 应用生命游戏规则更新细胞状态
    if (AT(x, y, z) && (alive_nei_num < 5 || alive_nei_num > 7))
        nextGrid[idx] = 0;
    else if (!AT(x, y, z) && alive_nei_num == 6)
        nextGrid[idx] = 1;
    else
        nextGrid[idx] = AT(x, y, z);
}

// 主函数，在 GPU 上运行 3D 生命游戏
void life3d_gpu(int N, char* grid, int T, int threadBlockSize) {
// 分配 GPU 内存
//定义需要的局部变量
    char *grid_c, *nextGrid;
    hipMalloc(&grid_c, N * N * N);
    hipMalloc(&nextGrid, N * N * N);
    // 将数据从主机复制到 GPU 内存
    hipMemcpy(grid_c, grid, N * N * N, hipMemcpyHostToDevice);
    // 定义线程块和线程块网格的大小
    dim3 threadsPerBlock(threadBlockSize, threadBlockSize, threadBlockSize); //thread x,y,z
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (N + threadsPerBlock.z - 1) / threadsPerBlock.z);
    // 迭代 T 次更新状态
    for (int t = 0; t < T; t++) {
        life3d_kernel<<<numBlocks, threadsPerBlock>>>(N, grid_c, nextGrid);
        hipDeviceSynchronize();
    EXCHANGE_PTR(grid_c,nextGrid);
    }
    // 将最终结果从 GPU 复制回主机
    hipMemcpy(grid, grid_c, N * N * N, hipMemcpyDeviceToHost);

    hipFree(grid_c);
    hipFree(nextGrid);
}

// 读取输入文件
// input_file 输入文件的路径
// buffer 存储读取数据的缓冲区
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
// output_file 输出文件的路径
// buffer 存储要写入数据的缓冲区
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
//定义需要的变量，包括分块尺寸等
    int threadBlockSize=4;					//块尺寸
    int N = std::stoi(argv[1]);					//世界网格大小
    int T = std::stoi(argv[2]);					//迭代次数
    char *input_file = argv[3];				//初始文件
    char *output_file = argv[4];				//输出文件
    char *grid = (char *)malloc(N * N * N);		// 分配主机内存存储网格数据
    read_file(input_file, grid);					// 从文件读取初始状态
    int start_pop = population(N, grid);			// 计算初始存活细胞的数量
    auto start_time = std::chrono::high_resolution_clock::now();		//统计时间end_time-start_time
    // 在 GPU 上运行 3D 生命游戏
    life3d_gpu(N, grid, T, threadBlockSize);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    // 计算最终存活细胞的数量
    int final_pop = population(N, grid);
    //将计算结果存入文件中
    write_file(output_file, grid, N);
    
    cout << "TPB:" << threadBlockSize << "*" << threadBlockSize << "*" << threadBlockSize << endl;
    cout << "GPU:" << endl;
    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;
    
    free(grid);
    return 0;
}

